#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/library_types.h>

#include "playground/cublas_handle.hpp"
#include "playground/matmul.hpp"
#include "playground/system.hpp"

namespace playground
{
PLAYGROUND_MATMUL_DEC(float16_t, 1, M, N, K, A, B, C)
{
    const float16_t Alpha = 1.0F;
    const float16_t Beta = 0.0F;
    hipblasGemmEx(s_getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
                 &Alpha, B, HIP_R_16F, N, A, HIP_R_16F, K, &Beta, C,
                 HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
}  // namespace playground
