#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "playground/cublas_handle.hpp"
#include "playground/matmul.hpp"
#include "playground/system.hpp"

namespace playground
{
PLAYGROUND_MATMUL_DEC(float32_t, 1, M, N, K, A, B, C)
{
    const float32_t Alpha = 1.0F;
    const float32_t Beta = 0.0F;
    hipblasSgemm(s_getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &Alpha,
                B, N, A, K, &Beta, C, N);
}
}  // namespace playground